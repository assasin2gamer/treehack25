#include "hip/hip_runtime.h"
// File: ica_processor.cu
#include "signal_processing.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>

// Error checking macros
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if(err != hipSuccess) { \
       std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << " code:" << err << " msg:" << hipGetErrorString(err) << std::endl; \
       exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t stat = call; \
    if(stat != HIPBLAS_STATUS_SUCCESS) { \
       std::cerr << "CUBLAS error in " << __FILE__ << ":" << __LINE__ << std::endl; \
       exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUSOLVER(call) { \
    hipsolverStatus_t stat = call; \
    if(stat != HIPSOLVER_STATUS_SUCCESS) { \
       std::cerr << "CUSOLVER error in " << __FILE__ << ":" << __LINE__ << std::endl; \
       exit(EXIT_FAILURE); \
    } \
}

// Kernel to compute mean per channel
__global__ void computeMeanKernel(const float* data, float* means, int n_samples, int n_channels) {
    extern __shared__ float sdata[];
    int j = blockIdx.x;
    int tid = threadIdx.x;
    float sum = 0.0f;
    for (int i = tid; i < n_samples; i += blockDim.x) {
        sum += data[i + j * n_samples];
    }
    sdata[tid] = sum;
    __syncthreads();
    for (int s = blockDim.x/2; s > 0; s /= 2) {
        if (tid < s) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        means[j] = sdata[0] / n_samples;
    }
}

// Kernel to subtract channel means
__global__ void subtractMeanKernel(float* data, const float* means, int n_samples, int n_channels) {
    int j = blockIdx.x;
    int i = threadIdx.x;
    if (j < n_channels && i < n_samples) {
        data[i + j * n_samples] -= means[j];
    }
}

// Kernel to compute nonlinearity and its derivative for fastICA
__global__ void computeGKernel(const float* u, float* gu, float* gprime, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float tanh_u = tanhf(u[idx]);
        gu[idx] = tanh_u;
        gprime[idx] = 1.0f - tanh_u * tanh_u;
    }
}

// Kernel for reduction sum using shared memory
__global__ void reduceSumKernel(const float* input, float* output, int n) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    for (int i = idx; i < n; i += blockDim.x * gridDim.x) {
        sum += input[i];
    }
    sdata[tid] = sum;
    __syncthreads();
    for (int s = blockDim.x/2; s > 0; s /= 2) {
        if (tid < s) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        atomicAdd(output, sdata[0]);
    }
}

void computeCovariance(hipblasHandle_t handle, float* d_X, float* d_cov, int n_samples, int n_channels) {
    float alpha = 1.0f / n_samples;
    float beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                             n_channels, n_channels, n_samples,
                             &alpha,
                             d_X, n_samples,
                             d_X, n_samples,
                             &beta,
                             d_cov, n_channels));
}

void computeWhiteningMatrix(hipsolverHandle_t cusolverH, hipblasHandle_t cublasH, float* d_cov, float* whitening, int n_channels) {
    int lwork = 0;
    int info_gpu = 0;
    float *d_work = nullptr;
    int *devInfo = nullptr;
    CHECK_CUDA(hipMalloc((void**)&devInfo, sizeof(int)));

    float *d_W = nullptr;
    CHECK_CUDA(hipMalloc((void**)&d_W, sizeof(float) * n_channels * n_channels));
    CHECK_CUDA(hipMemcpy(d_W, d_cov, sizeof(float) * n_channels * n_channels, hipMemcpyDeviceToDevice));

    float* d_eigenvalues = nullptr;
    CHECK_CUDA(hipMalloc((void**)&d_eigenvalues, sizeof(float) * n_channels));

    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
    CHECK_CUSOLVER(hipsolverDnSsyevd_bufferSize(cusolverH, jobz, uplo, n_channels, d_W, n_channels, d_eigenvalues, &lwork));
    CHECK_CUDA(hipMalloc((void**)&d_work, sizeof(float) * lwork));
    CHECK_CUSOLVER(hipsolverDnSsyevd(cusolverH, jobz, uplo, n_channels, d_W, n_channels, d_eigenvalues, d_work, lwork, devInfo));
    CHECK_CUDA(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (info_gpu != 0) {
        std::cerr << "Error: hipsolverDnSsyevd failed" << std::endl;
        exit(EXIT_FAILURE);
    }

    std::vector<float> h_eigenvalues(n_channels);
    CHECK_CUDA(hipMemcpy(h_eigenvalues.data(), d_eigenvalues, sizeof(float) * n_channels, hipMemcpyDeviceToHost));
    hipFree(d_eigenvalues);

    std::vector<float> h_V(n_channels * n_channels);
    CHECK_CUDA(hipMemcpy(h_V.data(), d_W, sizeof(float) * n_channels * n_channels, hipMemcpyDeviceToHost));

    std::vector<float> h_diag(n_channels * n_channels, 0.0f);
    for (int i = 0; i < n_channels; i++) {
        float inv_sqrt = 1.0f / sqrtf(h_eigenvalues[i] + 1e-5f);
        h_diag[i + i * n_channels] = inv_sqrt;
    }
    std::vector<float> h_temp(n_channels * n_channels, 0.0f);
    for (int i = 0; i < n_channels; i++) {
        for (int j = 0; j < n_channels; j++) {
            h_temp[i + j * n_channels] = h_V[i + j * n_channels] * h_diag[j + j * n_channels];
        }
    }
    std::vector<float> h_whitening(n_channels * n_channels, 0.0f);
    for (int i = 0; i < n_channels; i++) {
        for (int j = 0; j < n_channels; j++) {
            float sum = 0.0f;
            for (int k = 0; k < n_channels; k++) {
                sum += h_temp[i + k * n_channels] * h_V[j + k * n_channels];
            }
            h_whitening[i + j * n_channels] = sum;
        }
    }
    CHECK_CUDA(hipMemcpy(whitening, h_whitening.data(), sizeof(float) * n_channels * n_channels, hipMemcpyHostToDevice));

    hipFree(d_W);
    hipFree(d_work);
    hipFree(devInfo);
}

void fastICA(hipblasHandle_t cublasH, float* d_Y, int n_samples, int n_channels, float* d_W, int maxIter = 5, float tol = 1e-4f) {
    float* d_w;
    float* d_w_old;
    float* d_u;
    float* d_gu;
    float* d_gprime;
    float* d_temp;
    CHECK_CUDA(hipMalloc((void**)&d_w, sizeof(float) * n_channels));
    CHECK_CUDA(hipMalloc((void**)&d_w_old, sizeof(float) * n_channels));
    CHECK_CUDA(hipMalloc((void**)&d_u, sizeof(float) * n_samples));
    CHECK_CUDA(hipMalloc((void**)&d_gu, sizeof(float) * n_samples));
    CHECK_CUDA(hipMalloc((void**)&d_gprime, sizeof(float) * n_samples));
    CHECK_CUDA(hipMalloc((void**)&d_temp, sizeof(float) * n_channels));

    for (int comp = 0; comp < n_channels; comp++) {
        std::vector<float> h_w(n_channels);
        for (int i = 0; i < n_channels; i++) {
            h_w[i] = (float)rand()/RAND_MAX;
        }
        float norm = 0.0f;
        for (int i = 0; i < n_channels; i++) {
            norm += h_w[i]*h_w[i];
        }
        norm = sqrtf(norm);
        for (int i = 0; i < n_channels; i++) {
            h_w[i] /= norm;
        }
        CHECK_CUDA(hipMemcpy(d_w, h_w.data(), sizeof(float)*n_channels, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemset(d_w_old, 0, sizeof(float)*n_channels));

        for (int iter = 0; iter < maxIter; iter++) {
            CHECK_CUDA(hipMemcpy(d_w_old, d_w, sizeof(float)*n_channels, hipMemcpyDeviceToDevice));
            float alpha = 1.0f, beta = 0.0f;
            CHECK_CUBLAS(hipblasSgemv(cublasH, HIPBLAS_OP_N, n_samples, n_channels,
                                     &alpha, d_Y, n_samples, d_w, 1,
                                     &beta, d_u, 1));
            int blockSize = 256;
            int gridSize = (n_samples + blockSize - 1) / blockSize;
            computeGKernel<<<gridSize, blockSize>>>(d_u, d_gu, d_gprime, n_samples);
            hipDeviceSynchronize();
            float h_sum = 0.0f;
            float* d_sum;
            CHECK_CUDA(hipMalloc((void**)&d_sum, sizeof(float)));
            CHECK_CUDA(hipMemset(d_sum, 0, sizeof(float)));
            int redBlockSize = 256;
            int redGridSize = 32;
            reduceSumKernel<<<redGridSize, redBlockSize, redBlockSize * sizeof(float)>>>(d_gprime, d_sum, n_samples);
            hipDeviceSynchronize();
            CHECK_CUDA(hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost));
            float mean_gprime = h_sum / n_samples;
            hipFree(d_sum);
            beta = 0.0f;
            CHECK_CUBLAS(hipblasSgemv(cublasH, HIPBLAS_OP_T, n_samples, n_channels,
                                     &alpha, d_Y, n_samples, d_gu, 1,
                                     &beta, d_temp, 1));
            float scale = 1.0f / n_samples;
            CHECK_CUBLAS(hipblasSscal(cublasH, n_channels, &scale, d_temp, 1));
            float neg_mean = -mean_gprime;
            CHECK_CUBLAS(hipblasSaxpy(cublasH, n_channels, &neg_mean, d_w, 1, d_temp, 1));
            for (int j = 0; j < comp; j++) {
                float dot = 0.0f;
                CHECK_CUBLAS(hipblasSdot(cublasH, n_channels, d_temp, 1, d_W + j * n_channels, 1, &dot));
                float neg_dot = -dot;
                CHECK_CUBLAS(hipblasSaxpy(cublasH, n_channels, &neg_dot, d_W + j * n_channels, 1, d_temp, 1));
            }
            float norm_w = 0.0f;
            CHECK_CUBLAS(hipblasSnrm2(cublasH, n_channels, d_temp, 1, &norm_w));
            float inv_norm = 1.0f / norm_w;
            CHECK_CUBLAS(hipblasSscal(cublasH, n_channels, &inv_norm, d_temp, 1));
            float dot_w = 0.0f;
            CHECK_CUBLAS(hipblasSdot(cublasH, n_channels, d_temp, 1, d_w_old, 1, &dot_w));
            if (fabsf(fabsf(dot_w) - 1.0f) < tol) {
                CHECK_CUDA(hipMemcpy(d_w, d_temp, sizeof(float)*n_channels, hipMemcpyDeviceToDevice));
                break;
            }
            CHECK_CUDA(hipMemcpy(d_w, d_temp, sizeof(float)*n_channels, hipMemcpyDeviceToDevice));
        }
        CHECK_CUDA(hipMemcpy(d_W + comp * n_channels, d_w, sizeof(float)*n_channels, hipMemcpyDeviceToDevice));
    }

    hipFree(d_w);
    hipFree(d_w_old);
    hipFree(d_u);
    hipFree(d_gu);
    hipFree(d_gprime);
    hipFree(d_temp);
}

std::vector<std::vector<float>> performICA(const std::vector<std::vector<float>>& signals, int n_samples) {
    //std::cout << "ICA start";

    int n_channels = signals.size();
    std::vector<float> X(n_samples * n_channels, 0.0f);
    for (int ch = 0; ch < n_channels; ch++) {
        for (int i = 0; i < n_samples; i++) {
            X[i + ch * n_samples] = signals[ch][i];
        }
    }
    hipblasHandle_t cublasH;
    hipsolverHandle_t cusolverH;
    CHECK_CUBLAS(hipblasCreate(&cublasH));
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));

    float* d_X;
    CHECK_CUDA(hipMalloc((void**)&d_X, sizeof(float) * n_samples * n_channels));
    CHECK_CUDA(hipMemcpy(d_X, X.data(), sizeof(float) * n_samples * n_channels, hipMemcpyHostToDevice));
    //std::cout << "ICA 1";

    float* d_means;
    CHECK_CUDA(hipMalloc((void**)&d_means, sizeof(float) * n_channels));
    int blockSize = 256;
    int gridSize = n_channels;
    computeMeanKernel<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_X, d_means, n_samples, n_channels);
    hipDeviceSynchronize();
    subtractMeanKernel<<<gridSize, n_samples>>>(d_X, d_means, n_samples, n_channels);
    hipDeviceSynchronize();
    hipFree(d_means);
    //std::cout << "ICA 2";

    float* d_cov;
    CHECK_CUDA(hipMalloc((void**)&d_cov, sizeof(float) * n_channels * n_channels));
    computeCovariance(cublasH, d_X, d_cov, n_samples, n_channels);
    //std::cout << "ICA 3";

    float* d_whitening;
    CHECK_CUDA(hipMalloc((void**)&d_whitening, sizeof(float) * n_channels * n_channels));
    computeWhiteningMatrix(cusolverH, cublasH, d_cov, d_whitening, n_channels);
    hipFree(d_cov);
    //std::cout << "ICA 4";

    float* d_Y;
    CHECK_CUDA(hipMalloc((void**)&d_Y, sizeof(float) * n_samples * n_channels));
    float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             n_samples, n_channels, n_channels,
                             &alpha,
                             d_X, n_samples,
                             d_whitening, n_channels,
                             &beta,
                             d_Y, n_samples));
    hipFree(d_X);
    hipFree(d_whitening);
    //std::cout << "ICA 5";



    // Seems to die >50 components
    float* d_W;
    CHECK_CUDA(hipMalloc((void**)&d_W, sizeof(float) * n_channels * n_channels));
    fastICA(cublasH, d_Y, n_samples, n_channels, d_W);

    //std::cout << "ICA 7";

    float* d_S;
    CHECK_CUDA(hipMalloc((void**)&d_S, sizeof(float) * n_samples * n_channels));
    CHECK_CUBLAS(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             n_samples, n_channels, n_channels,
                             &alpha,
                             d_Y, n_samples,
                             d_W, n_channels,
                             &beta,
                             d_S, n_samples));
    hipFree(d_Y);
    hipFree(d_W);
    //std::cout << "ICA 6";
    std::vector<float> S(n_samples * n_channels);
    CHECK_CUDA(hipMemcpy(S.data(), d_S, sizeof(float) * n_samples * n_channels, hipMemcpyDeviceToHost));
    hipFree(d_S);

    hipblasDestroy(cublasH);
    hipsolverDnDestroy(cusolverH);

    std::vector<std::vector<float>> components(n_channels, std::vector<float>(n_samples));
    for (int ch = 0; ch < n_channels; ch++) {
        for (int i = 0; i < n_samples; i++) {
            components[ch][i] = S[i + ch * n_samples];
        }
    }
    //std::cout << "ICA complete";

    return components;
}
